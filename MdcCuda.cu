#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__
void mdcSequencialCUDA(int *x, int *y, int *resto)
{
    do
    {
        *resto = *x % *y;
    
        *x = *y;
        *y = *resto;
    }
    while (*resto != 0);
}

int main(void)
{
  // Iniciando a contagem da execução do algoritmo
  clock_t t; //variável para armazenar tempo
  t = clock(); //armazena tempo

  // hospedando cópias de x,y e resto
  int x, y, resto;
  // Cópias do sistema de x,y e resto
  int *d_x, *d_y, *d_resto;
  int size = sizeof(int);

  // Alocando espaco para as variáveis
  hipMalloc((void **) &d_x, size);
  hipMalloc((void **) &d_y, size);
  hipMalloc((void **) &d_resto, size);

  // Definido os valores de entrada
  x = 50;
  y = 20;
  resto = 1;

  // Copiando os valores de entrada para a GPU
  hipMemcpy(d_x, &x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, &y, size, hipMemcpyHostToDevice);
  hipMemcpy(d_resto, &resto, size, hipMemcpyHostToDevice);

  // Resolvendo a operacao MDC dentro da GPU
  mdcSequencialCUDA<<<1,1>>>(d_x, d_y, d_resto);

  // Copiando o resultado de volta para o host
  hipMemcpy(&x, d_x, size, hipMemcpyDeviceToHost);
  //cudaMemcpy(&y, d_y, size, cudaMemcpyDeviceToHost);
  //cudaMemcpy(&resto, d_resto, size, cudaMemcpyDeviceToHost);
  
  

  // Limpando a memória
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_resto);
  //free(x);
  //free(y);
  //free(resto);

  printf("Resultado: %i", x);

  // Finalizando a contagem da execução do algoritmo
  t = clock() - t; //tempo final - tempo inicial
  double tempo_execucao = (double)(((double)t)/(CLOCKS_PER_SEC/1000)); //Dando o resultado em milissegundos.
  //cout << "O tempo de execucao foi da aplicacao: " << (double)tempo_execucao << " milissegudos.";
  //printf("tempo de execucao: %f", tempo_execucao);
  printf(" Tempo gasto: %g ms.", tempo_execucao);

  

  return 0;

}
